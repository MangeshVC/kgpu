#include "hip/hip_runtime.h"
/* This work is licensed under the terms of the GNU GPL, version 2.  See
 * the GPL-COPYING file in the top-level directory.
 *
 * Copyright (c) 2010-2011 University of Utah and the Flux Group.
 * All rights reserved.
 *
 *
 * P and Q disk computing function, mostly derived from the kernel:
 * /lib/raid6/int.uc
 * Support x86_64 only.
 *
 * To be included by others.
 */

typedef unsigned long u64;
typedef unsigned char u8;

#define NBYTES(x) ((x) * 0x0101010101010101UL)
#define NSIZE  8
#define NSHIFT 3

#define SHLBYTE(v) (((v)<<1)&NBYTES(0xfe))
#define MASK(v) ({ u64 vv = (v)&NBYTES(0x80); (vv<<1)-(vv>>7);})

/*
 * @disks: number of disks, p and q included
 * @dsize: unit size, or a stripe?
 * @data: disk data 
 */
__global__ void raid6_pq(unsigned int disks, unsigned long dsize, u8 *data)
{
    u64 *d = (u64*)data;
    int z0, offset64, step64, tid;

    u64 wd0, wq0, wp0;
    
    tid = blockDim.x*blockIdx.x+threadIdx.x;
    step64 = dsize/sizeof(u64);
    z0 = disks-3;
    offset64 = step64*z0+tid;
    
    wq0 = wp0 = d[offset64];
    for (offset64 -= step64; offset64>=0; offset64 -=step64) {
	wd0 = d[offset64];
	wp0 ^= wd0;
	wq0 = SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ wd0;
    }
    d[step64*(z0+1)+tid] = wp0;
    d[step64*(z0+2)+tid] = wq0;    
}

/*
 * Fixed number of disks version
 * Naming: _fdx, where x is the number of disks, including p and q.
 *
 * shared memory seems not a necessary trick because every datum is
 * accessed only once.
 *
 */
__global__ void raid6_pq_fd6(unsigned int disks, unsigned long dsize, u8 *data)
{
    u64 *d;;
    int step64, tid;

    u64 wq0, wp0;

    __shared__ u64 dsk[4][THREADS_PER_BLOCK];

    tid = blockDim.x*blockIdx.x+threadIdx.x;
    step64 = dsize/sizeof(u64);
    d = ((u64*)data)+tid;
    
    dsk[0][threadIdx.x] = *d;
    d += step64;
    dsk[1][threadIdx.x] = *d;
    d += step64;
    dsk[2][threadIdx.x] = *d;
    d += step64;
    dsk[3][threadIdx.x] = *d;
    d += step64;
    
    wq0 = wp0 = dsk[3][threadIdx.x];
    
    wp0 ^= dsk[2][threadIdx.x];
    wq0 =
	SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ dsk[2][threadIdx.x];
    wp0 ^= dsk[1][threadIdx.x];
    wq0 =
	SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ dsk[1][threadIdx.x];
    wp0 ^= dsk[0][threadIdx.x];
    wq0 =
	SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ dsk[0][threadIdx.x];
    
    *d = wp0;
    *(d+step64) = wq0;
}
