
#include <hip/hip_runtime.h>
/* This work is licensed under the terms of the GNU GPL, version 2.  See
 * the GPL-COPYING file in the top-level directory.
 *
 * Copyright (c) 2010-2011 University of Utah and the Flux Group.
 * All rights reserved.
 *
 *
 * P and Q disk computing function, mostly derived from the kernel:
 * /lib/raid6/int.uc
 * Support x86_64 only.
 *
 * To be included by others.
 */

typedef unsigned long u64;
typedef unsigned char u8;

#define NBYTES(x) ((x) * 0x0101010101010101UL)
#define NSIZE  8
#define NSHIFT 3

#define SHLBYTE(v) (((v)<<1)&NBYTES(0xfe))
#define MASK(v) ({ u64 vv = (v)&NBYTES(0x80); (vv<<1)-(vv>>7);})

/*
 * @disks: number of disks, p and q included
 * @dsize: unit size, or a stripe?
 * @data: disk data 
 */
__global__ void raid6_pq(unsigned int disks, unsigned long dsize, u8 *data)
{
    u64 *d = (u64*)data;
    int z0, offset64, step64, tid;

    u64 wd0, wq0, wp0;
    
    tid = blockDim.x*blockIdx.x+threadIdx.x;
    step64 = dsize/sizeof(u64);
    z0 = disks-3;
    offset64 = step64*z0+tid;
    
    wq0 = wp0 = d[offset64];
    for (offset64 -= step64; offset64>=0; offset64 -=step64) {
	wd0 = d[offset64];
	wp0 ^= wd0;
	wq0 = SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ wd0;
    }
    d[step64*(z0+1)+tid] = wp0;
    d[step64*(z0+2)+tid] = wq0;    
}

/*
 * Fixed number of disks version
 * Naming: _fdx, where x is the number of disks, including p and q.
 *
 */
__global__ void raid6_pq_fd6(unsigned int disks, unsigned long dsize, u8 *data)
{
    u64 *d;;
    int step64, tid;

    u64 wq0, wp0;

    tid = blockDim.x*blockIdx.x+threadIdx.x;
    step64 = dsize/sizeof(u64);
    d = ((u64*)data)+tid+3*step64;
    
    wq0 = wp0 = *d;
    d -= step64;
    
    wp0 ^= *d;
    wq0 =
	SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ *d;
    d-= step64;
    
    wp0 ^= *d;
    wq0 =
	SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ *d;
    d -= step64;
    
    wp0 ^= *d;
    wq0 =
	SHLBYTE(wq0) ^ (MASK(wq0)&NBYTES(0x1d)) ^ *d;
    d += 4*step64;
    
    *d = wp0;
    *(d+step64) = wq0;
}
