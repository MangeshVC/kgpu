#include <hip/hip_runtime.h>
#include <stdio.h>
#include "nsk.h"
#include "devutils.h"

int devmemuses[3];

volatile void* dh_mems[3];
volatile void *h_mems[4];

hipStream_t ss[3];

void alloc_hdmem(void **pph, void **ppd, unsigned int size, mem_mode_t memMode)
{
    switch(memMode) {
    case PINNED:
	csc( hipHostAlloc(pph, size, 0) );
	if (ppd != NULL)
	    csc( hipMalloc(ppd, size) );
	break;
    case PAGEABLE:
	*pph = malloc(size);
	if (ppd != NULL)
	    csc( hipMalloc(ppd, size) );
	break;
    case MAPPED:
	csc( hipHostAlloc(pph, size, hipHostMallocMapped) );
	csc( hipHostGetDevicePointer(ppd, *pph, 0) );
	break;
    case WC:
	csc( hipHostAlloc(pph, size,
			   hipHostMallocMapped|hipHostMallocWriteCombined) );
	csc( hipHostGetDevicePointer(ppd, *pph, 0) );
    default:
	break;
    }
}

void free_hdmem(void **pph, void **ppd, mem_mode_t memMode)
{
    switch(memMode) {
    case PINNED:
    case MAPPED:
    case WC:
	csc(hipHostFree(*pph));
	break;
    case PAGEABLE:
	free(*pph);
    default:
	break;
    }

    *pph = NULL;
    if (ppd != NULL)
    	csc(hipFree(*ppd));
    *ppd = NULL;
}

void _csc(hipError_t e, const char *file, int line)
{
    if (e != hipSuccess){
	printf("nsk Error: %s %d %s\n", file, line, hipGetErrorString(e));
	hipDeviceReset();
	exit(0);
    }
}

void init_hd_buffers()
{
    hipStream_t s = ss[SKERNEL];
    // init device memory buffers uses:
    for (int i=0; i<3; i++)
	devmemuses[i] = -1; // nobody use
	
    // allocate memory buffers for data input and output
    for (int i=0; i<3; i++) {
	ALLOC_HDMEM(&(h_mems[i]), &(dh_mems[i]), NSK_MEM_SIZE, PINNED);
    }
    csc( hipHostAlloc( (void**)&(h_mems[3]), NSK_MEM_SIZE, PINNED ) );

    for (int i=0; i<4; i++) {
	memset((void*)h_mems[i], 0, NSK_MEM_SIZE);
	if (i!= 3)
	    csc( h2d_cpy_a( dh_mems[i], h_mems[i], NSK_MEM_SIZE, s ) );
    }
    csc( hipStreamSynchronize(s) );
}

void init_hd_streams()
{
    for (int i=0; i<3; i++)
	csc( hipStreamCreate(&ss[i]) );
}

volatile void* get_next_device_mem(int user)
{
    int i;

    for (i=0; i<3; i++) {
	if (devmemuses[i] == -1) {
	    devmemuses[i] = user;
	    return dh_mems[i];
	}
    }
    
    return NULL;
}

void put_device_mem(volatile void* devmem)
{
    int i;

    for (i=0; i<3; i++) {
	if (devmem == dh_mems[i]) {
	    devmemuses[i] = -1;
	    return;
	}
    }
}
