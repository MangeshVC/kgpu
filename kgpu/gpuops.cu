#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "helper.h"
#include "gputils.h"

extern "C" void init_gpu();
extern "C" void finit_gpu();

extern "C" void *alloc_pinned_mem(unsigned long size);
extern "C" void free_pinned_mem(void *p);

extern "C" int alloc_gpu_mem(struct service_request *sreq);
extern "C" void free_gpu_mem(struct service_request *sreq);
extern "C" int alloc_stream(struct service_request *sreq);
extern "C" void free_stream(struct service_request *sreq);
//extern "C" struct service_request* alloc_service_request();
//extern "C" void free_service_request(struct service_request *sreq);

extern "C" int execution_finished(struct service_request *sreq);
extern "C" int post_finished(struct service_request *sreq);

#define MAX_STREAM_NR 4
static hipStream_t streams[MAX_STREAM_NR];
static int streamuses[MAX_STREAM_NR];

static const dim3 default_block_size(32,1);
static const dim3 default_grid_size(512,1);

static struct gpu_buffer devbufs[KGPU_BUF_NR];
static int devbufuses[KGPU_BUF_NR];

void init_gpu()
{
    int i;

    for (i=0; i< KGPU_BUF_NR; i++) {
	devbufs[i].addr = alloc_dev_mem(KGPU_BUF_SIZE);
	devbufs[i].size = KGPU_BUF_SIZE;
	devbufuses[i] = 0;
    }

    for (i=0; i<MAX_STREAM_NR; i++) {
        csc( hipStreamCreate(&streams[i]) );
	streamuses[i] = 0;
    }
}

void finit_gpu()
{
    int i;

    for (i=0; i<KGPU_BUF_NR; i++) {
	free_dev_mem(devbufs[i].addr);
    }
    for (i=0; i<MAX_STREAM_NR; i++) {
	csc( hipStreamDestroy(streams[i]));
    }
}

static hipStream_t get_stream(int stid)
{
    if (stid < 0 || stid >= MAX_STREAM_NR)
	return 0;
    else
	return streams[stid];
}

void *alloc_pinned_mem(unsigned long size) {
    void *h;
    csc( hipHostAlloc(&h, size, 0) );
    return h;
}

void free_pinned_mem(void* p) {
    csc( hipHostFree(p) );
}

static int __check_stream_done(hipStream_t s)
{
    hipError_t e = hipStreamQuery(s);
    if (e == hipSuccess) {
	return 1;
    } else if (e != hipErrorNotReady)
	csc(e);

    return 0;
}

int execution_finished(struct service_request *sreq)
{
    hipStream_t s = get_stream(sreq->stream_id);
    return __check_stream_done(s);
}

int post_finished(struct service_request *sreq)
{
    hipStream_t s = get_stream(sreq->stream_id);
    return __check_stream_done(s);
}

int alloc_gpu_mem(struct service_request *sreq)
{
    int i;

    for (i=0; i<KGPU_BUF_NR; i++) {
	if (!devbufuses[i]) {
	    devbufuses[i] = 1;
	    sreq->dinput = devbufs[i].addr;
	    sreq->doutput = (void*)(
		(unsigned long)(sreq->dinput)
		+ 256*((sreq->kureq.insize/256)? (sreq->kureq.insize/256+1):sreq->kureq.insize/256));
	    return 0;
	}
    }
    return 1;
}

void free_gpu_mem(struct service_request *sreq)
{
    int i;

    for (i=0; i<KGPU_BUF_NR; i++) {
	if (sreq->dinput == devbufs[i].addr) {
	    devbufuses[i] = 0;
	    sreq->dinput = NULL;
	    sreq->doutput = NULL;
	}
    }
}

int alloc_stream(struct service_request *sreq)
{
    int i;

    for (i=0; i<MAX_STREAM_NR; i++) {
	if (!streamuses[i]) {
	    streamuses[i] = 1;
	    sreq->stream_id = i;	    
	    return 0;
	}
    }
    return 1;
}

void free_stream(struct service_request *sreq)
{
    if (sreq->stream_id >= 0 && sreq->stream_id < MAX_STREAM_NR) {
	streamuses[sreq->stream_id] = 0;
    }
}


int default_compute_size(struct service_request *sreq)
{
    sreq->block_x = default_block_size.x;
    sreq->block_y = default_block_size.y;
    sreq->grid_x = default_grid_size.x;
    sreq->grid_y = default_grid_size.y;
    return 0;
}

int default_prepare(struct service_request *sreq)
{
    hipStream_t s = get_stream(sreq->stream_id);
    csc( ah2dcpy( sreq->dinput, sreq->kureq.input, sreq->kureq.insize, s) );
    return 0;
}

int default_post(struct service_request *sreq)
{
    hipStream_t s = get_stream(sreq->stream_id);
    csc( ad2hcpy( sreq->kureq.output, sreq->doutput, sreq->kureq.outsize, s) );
    return 0;
}
